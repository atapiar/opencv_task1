#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>

using namespace cv;
using namespace std;

__global__ void templateMatching(const cv::cuda::PtrStepSz<float> img, const cv::cuda::PtrStepSz<float> templ, cv::cuda::PtrStepSz<float> result) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= img.cols - templ.cols || y >= img.rows - templ.rows)
        return;

    float sum = 0.0f;
    for (int i = 0; i < templ.rows; ++i) {
        for (int j = 0; j < templ.cols; ++j) {
            sum += powf(img.ptr(y + i)[x + j] - templ.ptr(i)[j], 2);
        }
    }
    result.ptr(y)[x] = sum;
}

int main() {
    // Load input image and template
    Mat img = imread("input_image.jpg", IMREAD_GRAYSCALE);
    Mat templ = imread("template_image.jpg", IMREAD_GRAYSCALE);

    if (img.empty() || templ.empty()) {
        cerr << "Error: Couldn't load image(s)." << endl;
        return -1;
    }

    // Convert input image and template to float
    img.convertTo(img, CV_32F);
    templ.convertTo(templ, CV_32F);

    // Allocate memory on GPU
    cuda::GpuMat gpuImg(img);
    cuda::GpuMat gpuTempl(templ);
    cuda::GpuMat gpuResult(img.rows - templ.rows + 1, img.cols - templ.cols + 1, CV_32F);

    // Define block and grid dimensions
    dim3 block(16, 16);
    dim3 grid((img.cols - templ.cols + block.x - 1) / block.x, (img.rows - templ.rows + block.y - 1) / block.y);

    // Perform template matching on GPU
    templateMatching<<<grid, block>>>(gpuImg, gpuTempl, gpuResult);

    // Download result from GPU
    Mat result;
    gpuResult.download(result);

    // Find minimum and maximum values in the result
    double minVal, maxVal;
    Point minLoc, maxLoc;
    minMaxLoc(result, &minVal, &maxVal, &minLoc, &maxLoc);

    // Display the result
    rectangle(img, maxLoc, Point(maxLoc.x + templ.cols, maxLoc.y + templ.rows), Scalar::all(255), 2);
    imshow("Result", img);
    waitKey(0);

    return 0;
}
